#include "hip/hip_runtime.h"
#include "transform.cuh"
#include <malloc.h>
#include <assert.h>
#include <stdio.h>
#include "operation.h"
#include "cuRPCF.h"
#include <omp.h>
#include "transpose.cuh"
#include <iostream>

hipfftHandle planXYr2c, planXYc2r, planZ_pad, planZ_no_pad;
hipfftHandle planXYr2c_X3, planXYc2r_X6, planZ_X6, planZ_X3;

#define KERNEL_SYNCHRONIZED

hipEvent_t start_trans, end_trans;

__host__ int initFFT(problem &pb) {
	hipfftResult res;
	const int mx = pb.mx;
	const int my = pb.my;
	const int mz = pb.mz;
	const int inPitch = pb.pitch;
	const int outPitch = pb.tPitch;
	const int pmx = inPitch / sizeof(real);
	const int pmz = outPitch / sizeof(complex);
	const int nx = mx  / 3 * 2;
	const int ny = my  / 3 * 2;

	const int istride = 1;
	int inembed[2] = { my, pmx };
	int idist = pmx*my;

	int inembed2[2] = { my,pmx / 2 };

	int idist2 = pmx / 2 * my;

	int dim2[2] = { my,mx };

	int dim1[1] = { mz };
	int onembed[1] = { pmz };
	const int odist = pmz;
	const int ostride = 1;

	int dim1_no_pad[1] = { mz / 2 };

	//hipfftPlanMany( plan *, int dim, int* n, int* inembed, int istride, int idist
	//  int* onembed, int ostride, int odist, hipfftType, int batch);
	res = hipfftPlanMany(&planXYr2c, 2, dim2, inembed, istride, idist,
		inembed2, istride, idist2, myCUFFT_R2C, pb.pz);
	assert(res == HIPFFT_SUCCESS);
	res = hipfftPlanMany(&planXYc2r, 2, dim2, inembed2, istride, idist2,
		inembed, istride, idist, myCUFFT_C2R, pb.pz);
	assert(res == HIPFFT_SUCCESS);
	res = hipfftPlanMany(&planZ_pad, 1, dim1, onembed, ostride, odist,
		onembed, ostride, odist, myCUFFT_C2C, (nx/2+1)*ny);
	assert(res == HIPFFT_SUCCESS);
	res = hipfftPlanMany(&planZ_no_pad, 1, dim1_no_pad, onembed, ostride, odist,
		onembed, ostride, odist, myCUFFT_C2C, (nx/2+1)*ny);
	assert(res == HIPFFT_SUCCESS);

	//res = hipfftPlanMany(&planXYr2c_X3, 2, dim2, inembed, istride, idist,
	//	inembed2, istride, idist2, myCUFFT_R2C, pb.pz*3);
	//assert(res == HIPFFT_SUCCESS);
	//res = hipfftPlanMany(&planXYc2r_X6, 2, dim2, inembed2, istride, idist2,
	//	inembed, istride, idist, myCUFFT_C2R, pb.pz*6);

	//res = hipfftPlanMany(&planZ_X3, 1, dim1, onembed, ostride, odist,
	//	onembed, ostride, odist, myCUFFT_C2C, (nx / 2 + 1)*ny*3);
	//res = hipfftPlanMany(&planZ_X6, 1, dim1, onembed, ostride, odist,
	//	onembed, ostride, odist, myCUFFT_C2C, (nx / 2 + 1)*ny * 6);


	assert(res == HIPFFT_SUCCESS);

	assert(res == HIPFFT_SUCCESS);

	hipEventCreate(&start_trans);
	hipEventCreate(&end_trans);

	return 0;
}

__host__ int transform_3d_one(DIRECTION dir, hipPitchedPtr& Ptr,
	hipPitchedPtr& tPtr, int* dim, int* tDim, 
	Padding_mode pd, bool isOutput) {

	//transform in x-y direction
	hipfftResult res;

	hipExtent extent = make_hipExtent(
	  2*(dim[0]/2+1) * sizeof(real), dim[1], dim[2]);
	hipError_t err;

	ASSERT(dim[0] == tDim[1]);
	ASSERT(dim[1] == tDim[2]);
	ASSERT(dim[2] == tDim[0]);

	int nx = dim[0]  / 3 * 2;
	int ny = dim[1]  / 3 * 2;

	hipExtent tExtent = make_hipExtent(
		tDim[0] * sizeof(complex), nx/2+1 , ny);
	hipExtent pExtent = make_hipExtent(
		2 * (dim[0] / 2 + 1) * sizeof(real), dim[1], dim[2]/2+1);

	dim3 threadDim(4, 4);

//	real* buffer;
//	real* tbuffer;
	float time;
	// tPtr -> Ptr
	if (dir == BACKWARD) {

//		size_t size = Ptr.pitch*dim[1] * dim[2];
//		size_t pSize = Ptr.pitch*dim[1] * (dim[2]/2+1);
//		size_t tSize = tPtr.pitch*(nx / 2 + 1)*ny;
//		buffer = (real*)malloc(size);
//		tbuffer = (real*)malloc(tSize);
//		ASSERT(buffer != nullptr);
//		ASSERT(tbuffer != nullptr);

		//setZeros <<<1, threadDim >>> (Ptr, dim[0], dim[1], dim[2]);

//#ifdef DEBUG
//		err = hipMemcpy(tbuffer, tPtr.ptr, tSize, hipMemcpyDeviceToHost);
//		ASSERT(err == hipSuccess);
//		err = hipDeviceSynchronize();
//		ASSERT(err == hipSuccess);
//		if(isOutput) RPCF::write_3d_to_file("beforeREV.txt", tbuffer, tPtr.pitch, 2 * dim[2], (dim[0] / 2 + 1), dim[1]);
//#endif //DEBUG
		//chebyshev transform in z direction
		cheby_s2p(tPtr, dim[0] / 2 + 1, dim[1] , dim[2]);

		//transpose(dir, Ptr, tPtr, dim, tDim);
		hipEventRecord(start_trans);
		cuCheck(myCudaMalloc(Ptr, XYZ_3D), "my hipMalloc");
		cuda_transpose(dir, Ptr, tPtr, dim, tDim);
		cuCheck(myCudaFree(tPtr, ZXY_3D), "my cuda free at transform");

		hipEventRecord(end_trans);
		hipEventSynchronize(end_trans);
		hipEventElapsedTime(&time, start_trans, end_trans);
		std::cout << "transpose backward time = " << time / 1000.0 << std::endl;

		hipEventRecord(start_trans);
		
		setZeros((complex*)Ptr.ptr, Ptr.pitch, dim3(dim[0], dim[1], dim[2]));

		hipEventRecord(end_trans);
		hipEventSynchronize(end_trans);
		hipEventElapsedTime(&time, start_trans, end_trans);
		std::cout << "set zeros time = " << time / 1000.0 << std::endl;

		hipEventRecord(start_trans);
		void* dev_buffer = get_fft_buffer_ptr();
		res = CUFFTEXEC_C2R(planXYc2r, (CUFFTCOMPLEX*)Ptr.ptr,
			(CUFFTREAL*)Ptr.ptr);
			//(CUFFTREAL*)dev_buffer);
		//cuCheck(hipMemcpy(Ptr.ptr, dev_buffer, pSize, hipMemcpyDeviceToDevice),"mem move");

		hipEventRecord(end_trans);
		hipEventSynchronize(end_trans);
		hipEventElapsedTime(&time, start_trans, end_trans);
		std::cout << "FFT XY BACKWARD TIME = " << time / 1000.0 << std::endl;

		ASSERT(res == HIPFFT_SUCCESS);
		err = hipDeviceSynchronize();
		ASSERT(err == hipSuccess);

//#ifdef DEBUG
//		err = hipMemcpy(buffer, Ptr.ptr, size, hipMemcpyDeviceToHost);
//		ASSERT(err == hipSuccess);
//		err = hipDeviceSynchronize();
//		ASSERT(err == hipSuccess);
//		if (isOutput) RPCF::write_3d_to_file("afterREV.txt", buffer, Ptr.pitch, 2 * (dim[0] / 2 + 1), dim[1], dim[2]);
//#endif //DEBUG


//#ifdef DEBUG
//		err = hipMemcpy(buffer, Ptr.ptr, size, hipMemcpyDeviceToHost);
//		ASSERT(err == hipSuccess);
//		err = hipDeviceSynchronize();
//		ASSERT(err == hipSuccess);
//		if (isOutput) RPCF::write_3d_to_file("afterNORM.txt", buffer, Ptr.pitch, 2 * (dim[0] / 2 + 1), dim[1], dim[2]);
//#endif //DEBUG

	}
	else
	{
		// Ptr -> tPtr

//		size_t size = Ptr.pitch*dim[1] * dim[2];
//		size_t pSize = Ptr.pitch*dim[1] * (dim[2] / 2 + 1);
//		size_t tSize = tPtr.pitch*(dim[0] / 2 + 1)*dim[1];
//		buffer = (real*)malloc(size);
//		tbuffer = (real*)malloc(tSize);
//		ASSERT(buffer != nullptr);
//		ASSERT(tbuffer != nullptr);

		//ASSERT(err == hipSuccess);

//#ifdef DEBUG
//		err = hipMemcpy(buffer, Ptr.ptr, size, hipMemcpyDeviceToHost);
//		ASSERT(err == hipSuccess);
//		err = hipDeviceSynchronize();
//		ASSERT(err == hipSuccess);
//		if (isOutput) RPCF::write_3d_to_file("before.txt", buffer, Ptr.pitch, 2*(dim[0]/2+1), dim[1], dim[2]);
//#endif //DEBUG

		ASSERT(dir == FORWARD);
		void* dev_buffer = get_fft_buffer_ptr();
		hipEventRecord(start_trans);
		
		res = CUFFTEXEC_R2C(planXYr2c, (CUFFTREAL*)Ptr.ptr,
			(CUFFTCOMPLEX*)Ptr.ptr); 
		
		hipEventRecord(end_trans);
		hipEventSynchronize(end_trans);
		hipEventElapsedTime(&time, start_trans, end_trans);
		std::cout << "FFT XY forward TIME = " << time / 1000.0 << std::endl;
			//(CUFFTCOMPLEX*)dev_buffer);
		//cuCheck(hipMemcpy(Ptr.ptr, dev_buffer, pSize, hipMemcpyDeviceToDevice), "mem move");
//#ifdef DEBUG
//		err = hipMemcpy(buffer, Ptr.ptr, size, hipMemcpyDeviceToHost);
//		ASSERT(err == hipSuccess);
//		err = hipDeviceSynchronize();
//		ASSERT(err == hipSuccess);
//
//		if (isOutput) RPCF::write_3d_to_file("afterXY.txt", buffer, Ptr.pitch, 2 * (dim[0] / 2 + 1), dim[1], dim[2]);
//#endif // DEBUG

		err = hipDeviceSynchronize();
		ASSERT(err == hipSuccess);
		hipEventRecord(start_trans);
		
		normalize(Ptr, dim3(dim[0], dim[1], dim[2]), 1.0 / dim[0] / dim[1]);

		hipEventRecord(end_trans);
		hipEventSynchronize(end_trans);
		hipEventElapsedTime(&time, start_trans, end_trans);
		std::cout << "normalize TIME = " << time / 1000.0 << std::endl;

		//transpose(FORWARD, Ptr, tPtr, dim, tDim);
		hipEventRecord(start_trans);
		cuCheck(myCudaMalloc(tPtr, ZXY_3D), "my hipMalloc");
		cuda_transpose(dir, Ptr, tPtr, dim, tDim);
		cuCheck(myCudaFree(Ptr, XYZ_3D), "my cuda free at transform");

		err = hipDeviceSynchronize();
		ASSERT(err == hipSuccess);

		hipEventRecord(end_trans);
		hipEventSynchronize(end_trans);
		hipEventElapsedTime(&time, start_trans, end_trans);
		std::cout << "tranpose forward TIME = " << time / 1000.0 << std::endl;

		//err = hipMemcpy(tbuffer, tPtr.ptr, tSize, hipMemcpyDeviceToHost);
		//ASSERT(err == hipSuccess);
		//err = hipDeviceSynchronize();
		//ASSERT(err == hipSuccess);

//#ifdef DEBUG
//		if (isOutput) RPCF::write_3d_to_file("Transposed.txt", tbuffer, tPtr.pitch, 2 * dim[2], (dim[0] / 2 + 1), dim[1]);
//#endif //DEBUG

		//transform in z direction
		cheby_p2s(tPtr, dim[0] / 2 + 1, dim[1], dim[2]);

//#ifdef DEBUG
//		err = hipMemcpy(tbuffer, tPtr.ptr, tSize, hipMemcpyDeviceToHost);
//		ASSERT(err == hipSuccess);
//		err = hipDeviceSynchronize();
//		ASSERT(err == hipSuccess);
//		if (isOutput) RPCF::write_3d_to_file("afterZ.txt", tbuffer, tPtr.pitch, 2 * dim[2], (dim[0] / 2 + 1), dim[1]);
//#endif //DEBUG

		//setZeros<<<1, threadDim >>>(Ptr, dim[0], dim[1], dim[2]);
		//err = hipDeviceSynchronize();
		//ASSERT(err == hipSuccess);
		
	}
//	free(buffer);
//	free(tbuffer);
	return 0;
}

__host__ int transform(DIRECTION dir, problem& pb) {
	int indim[3];
	int outdim[3];

	indim[0] = pb.mx;
	indim[1] = pb.my;
	indim[2] = pb.mz;

	outdim[0] = pb.mz;
	outdim[1] = pb.mx;
	outdim[2] = pb.my;

	if (dir == BACKWARD) {
		//transform_backward_X6(pb);
		//return 0;
		transform_3d_one(BACKWARD, pb.dptr_u, pb.dptr_tu, indim, outdim, Padding);
		transform_3d_one(BACKWARD, pb.dptr_v, pb.dptr_tv, indim, outdim, Padding);
		transform_3d_one(BACKWARD, pb.dptr_w, pb.dptr_tw, indim, outdim, Padding);
		transform_3d_one(BACKWARD, pb.dptr_omega_x, pb.dptr_tomega_x, indim, outdim, Padding);
		transform_3d_one(BACKWARD, pb.dptr_omega_y, pb.dptr_tomega_y, indim, outdim, Padding);
		transform_3d_one(BACKWARD, pb.dptr_omega_z, pb.dptr_tomega_z, indim, outdim, Padding);
	}
	if (dir == FORWARD) {
		//transform_forward_X3(pb);
		//return 0;
		transform_3d_one(FORWARD, pb.dptr_lamb_z, pb.dptr_tLamb_z, indim, outdim);
		transform_3d_one(FORWARD, pb.dptr_lamb_y, pb.dptr_tLamb_y, indim, outdim);
		transform_3d_one(FORWARD, pb.dptr_lamb_x, pb.dptr_tLamb_x, indim, outdim);
	}
	return 0;
}

//nx, ny, nz is the size of large matrix
//mx, my, mz is the size of the small matrix (dealiased)
__global__ void setZerosKernel(complex* ptr,size_t pitch, int mx, int my, int mz) {
	int ky = threadIdx.x + blockIdx.x*blockDim.x;
	int kz = threadIdx.y + blockIdx.y*blockDim.y;
	if (ky >= my || kz >= mz/2+1) return;
	size_t inc = pitch * (kz * my + ky)/sizeof(complex);
	ptr = ptr + inc;
	int nx = mx / 3 * 2;
	int ny = my / 3 * 2;
	
	if (ky >= ny / 2 && ky < my - (ny/2-1)) {
		for (int ix = 0; ix<mx/2+1; ix++) {
			ptr[ix] = 0.0;
		}
	}
	else
	{
		for (int ix = nx/2-1; ix<mx/2+1; ix++) {
			ptr[ix] = 0.0;
		}
	}
}

__host__ void setZeros(complex* ptr, size_t pitch, dim3 dims) {
	int nThreadx = 16;
	int nThready = 16;
	dim3 nThread(nThreadx, nThready);
	int dim[3] = { dims.x,dims.y,dims.z };
	int nDimx = dim[1] / nThreadx;
	int nDimy = (dim[2] / 2 + 1) / nThready;
	if (dim[1] % nThreadx != 0) nDimx++;
	if ((dim[2] / 2 + 1) % nThready != 0) nDimy++;
	dim3 nDim(nDimx, nDimy);
	setZerosKernel <<<nDim, nThread >>>((complex*)ptr, pitch,
		dim[0], dim[1], dim[2]);
#ifdef KERNEL_SYNCHRONIZED
	cuCheck(hipDeviceSynchronize(), "set zeros");
#endif
}

__global__ void normalizeKernel(hipPitchedPtr p, int mx, int my, int mz, real factor) {
	const int iy = blockIdx.x;
	const int iz = blockIdx.y;
	const int ix = threadIdx.x;
	if (iy >= my || iz >= mz/2+1)return;
	//const int ny = my / 3 * 2;
	//if (iy > ny / 2  && iy < my - (ny/2)) return;
	if (ix >= mx) return;

	size_t pitch = p.pitch; 
	size_t dist = pitch*(my*iz + iy) / sizeof(real);

	real* row = ((real*)p.ptr) + dist;
	row[ix] = row[ix] * factor;
}

__host__ void normalize(hipPitchedPtr Ptr, dim3 dims, real factor) {
	hipError_t err;
	int dim[3] = { dims.x,dims.y,dims.z }; 
	dim3 nDim(dim[1], dim[2] / 2 + 1);
	normalizeKernel<<<nDim, dim[0]>>> (Ptr, dim[0], dim[1], dim[2], factor);
#ifdef KERNEL_SYNCHRONIZED
	err = hipDeviceSynchronize();
#endif
	ASSERT(err == hipSuccess);
}


//preprocessing of chebyshev transform, spect to phy
__global__ void cheby_pre_s2p_pad(complex* u, const size_t pitch, const int hmx, const int my, const int mz) {
	const int mx = (hmx-1)*2;
	const int pz = mz / 2 + 1;
	const int nz = mz / 4;	//here, nz is the max index of z (start from 0)
	const int hnx = mx / 3 * 2 / 2 + 1;
	const int ny = my / 3 * 2;
	const int ix = blockIdx.x;
	const int iy = blockIdx.y;
	if (ix >= hnx || iy >= ny)return;
	const int iz = threadIdx.x;
	if (iz > nz)return;

	size_t dist = pitch*(hnx*iy + ix) / sizeof(complex);
	u = u + dist;
	/*for (int i = nz; i < pz; i++) {
		u[i].re = 0.0;
		u[i].im = 0.0;
	}*/
	u[iz + nz + 1] = 0.0;
	u[iz + pz - 1] = 0.0;
	/*for (int i = 0; i < nz; i++) {
		u[i].re = u[i].re*0.5;
		u[i].im = u[i].im*0.5;
	}*/
	u[iz] = u[iz] * 0.5;

	/*for (int i = 1; i < pz - 1; i++) {
		u[mz - i].re = u[i].re;
		u[mz - i].im = u[i].im;
	}*/

	if (iz == 0) {
		u[0] = u[0] * 2.0;
	}
	else {
		u[mz - iz] = u[iz];
	}
}

__global__ void cheby_pre_s2p_noPad(complex* u, const size_t pitch, const int hmx, const int my, const int mz) {
	const int mx = (hmx - 1) * 2;
	const int pz = mz / 2 + 1;
	const int nz = mz / 4;	//here, nz is the max index of z (start from 0)
	const int hnx = mx/ 3 * 2 / 2 + 1;
	const int ny = my / 3 * 2;
	const int ix = threadIdx.x + blockIdx.x*blockDim.x;
	const int iy = threadIdx.y + blockIdx.y*blockDim.y;
	if (ix >= hnx || iy >= ny)return;

	size_t dist = pitch*(hnx*iy + ix) / sizeof(complex);

	u = u + dist;
	//for (int i = nz; i < pz; i++) {
	//	u[i].re = 0.0;
	//	u[i].im = 0.0;
	//}
	for (int i = 0; i < nz; i++) {
		u[i].re = u[i].re*0.5;
		u[i].im = u[i].im*0.5;
	}
	for (int i = 1; i < nz - 1; i++) {
		u[pz-1 - i].re = u[i].re;
		u[pz-1 - i].im = u[i].im;
	}
	u[0].re = u[0].re*2.0;
	u[0].im = u[0].im*2.0;
}

//preprocessing of chebyshev transfor, phy to spect
__global__ void cheby_pre_p2s(complex* u, const size_t pitch, const int hmx, const int my, const int mz) {
	const int mx = (hmx - 1) * 2;
	const int pz = mz / 2 + 1;
	//	const int nz = mz / 4;	//here, nz is the max index of z (start from 0)
	const int hnx = mx / 3 * 2 / 2 + 1;
	const int ny = my / 3 * 2;
	const int ix = blockIdx.x;
	const int iy = blockIdx.y;
	const int iz = threadIdx.x;
	if (ix >= hnx || iy >= ny)return;
	if (iz >= pz - 1) return;
	if (iz == 0)return;
	size_t dist = pitch*(hnx*iy + ix) / sizeof(complex);
	u = u + dist;
	u[mz - iz].re = u[iz].re;
	u[mz - iz].im = u[iz].im;
}

__global__ void cheby_post_p2s(complex* u, const size_t pitch, const int hmx, const int my, const int mz) {
	const int mx = (hmx - 1) * 2;
	const int pz = mz / 2 + 1;
	//const int nz = mz / 4;	//here, nz is the max index of z (start from 0)
	const int hnx = mx/ 3 * 2 /2 + 1;
	const int ny = my / 3 * 2;
	const int ix = blockIdx.x;
	const int iy = blockIdx.y;
	if (ix >= hnx || iy >= ny)return;
	const int iz = threadIdx.x;
	if (iz >= pz)return;
	size_t dist = pitch*(hnx*iy + ix) / sizeof(complex);

	u = u + dist;
	real factor = (1.0 / (pz - 1));

	u[iz].re = u[iz].re*factor;
	u[iz].im = u[iz].im*factor;

	if (iz == 0) {
		u[0].re = u[0].re*0.5;
		u[0].im = u[0].im*0.5;
	}
}

__host__ void cheby_p2s(hipPitchedPtr tPtr, int hmx, int my, int mz) {
//	const size_t pitch = tPtr.pitch;
	const int nx = (hmx - 1) * 2 / 3 * 2;
	const int ny = my / 3 * 2;
	const int hnx = nx / 2 + 1;

	int threadDimx = 16;
	int threadDimy = 16;

	int blockDimx = hnx / threadDimx;
	int blockDimy = ny / threadDimy;

	if (hnx%threadDimx != 0) blockDimx++;
	if (ny%threadDimy != 0) blockDimy++;

	dim3 nthread(threadDimx, threadDimy);
	dim3 nBlock(blockDimx, blockDimy);

	hipfftResult res;
	hipError_t err;
	float time;
	hipEventRecord(start_trans);

	cheby_pre_p2s<<<dim3(hnx,ny),mz/2+1>>>((complex*)tPtr.ptr, tPtr.pitch, hmx, my, mz);
#ifdef KERNEL_SYNCHRONIZED
	err = hipDeviceSynchronize();
	assert(err == hipSuccess);
#endif
	hipEventRecord(end_trans);
	hipEventSynchronize(end_trans);
	hipEventElapsedTime(&time, start_trans, end_trans);
	std::cout << "cheby_pre_p2s_time = " << time / 1000.0 << std::endl;

	hipEventRecord(start_trans);
	res = CUFFTEXEC_C2C(planZ_pad, (CUFFTCOMPLEX*)tPtr.ptr,
		(CUFFTCOMPLEX*)tPtr.ptr, HIPFFT_FORWARD);
	assert(res == HIPFFT_SUCCESS);

	hipEventRecord(end_trans);
	hipEventSynchronize(end_trans);
	hipEventElapsedTime(&time, start_trans, end_trans);
	std::cout << "cheby fft p2s time = " << time / 1000.0 << std::endl;

	//err = hipDeviceSynchronize();
	//assert(err == hipSuccess);

	hipEventRecord(start_trans);
	cheby_post_p2s<<<dim3(hnx,ny),mz/2+1>>>((complex*)tPtr.ptr, tPtr.pitch, hmx, my, mz);
#ifdef KERNEL_SYNCHRONIZED
	err = hipDeviceSynchronize();
	assert(err == hipSuccess);
#endif
	hipEventRecord(end_trans);
	hipEventSynchronize(end_trans);
	hipEventElapsedTime(&time, start_trans, end_trans);
	std::cout << "cheby_post_p2s_time = " << time / 1000.0 << std::endl;
}
__host__ void cheby_s2p(hipPitchedPtr tPtr, int hmx, int my, int mz, Padding_mode doPadding) {
//	const size_t pitch = tPtr.pitch;
//	const int pz = mz / 2 + 1;
	const int nx = (hmx-1)*2/3*2;
	const int ny = my/3*2;
	const int hnx = nx/2+1;

	int threadDimx = 16;
	int threadDimy = 16;

	int blockDimx = hnx / threadDimx ;
	int blockDimy = ny / threadDimy ;

	if (hnx%threadDimx != 0) blockDimx++;
	if (ny%threadDimy != 0) blockDimy++;

	dim3 nthread(threadDimx, threadDimy);
	dim3 nBlock(blockDimx, blockDimy);
	hipfftResult res;
	hipError_t err;
	float time;

	if(doPadding == Padding){
		hipEventRecord(start_trans);
		cheby_pre_s2p_pad<<<dim3(hnx,ny), mz/4+1 >>>((complex*)tPtr.ptr, tPtr.pitch, hmx, my, mz);
#ifdef KERNEL_SYNCHRONIZED
		err = hipDeviceSynchronize();
		assert(err == hipSuccess);
#endif		
		hipEventRecord(end_trans);
		hipEventSynchronize(end_trans);
		hipEventElapsedTime(&time, start_trans, end_trans);
		std::cout << "cheby_pre_s2p_pad_time = " << time / 1000.0 << std::endl;


		hipEventRecord(start_trans);
		res = CUFFTEXEC_C2C(planZ_pad, (CUFFTCOMPLEX*)tPtr.ptr,
			(CUFFTCOMPLEX*)tPtr.ptr, HIPFFT_FORWARD);
		ASSERT(res == HIPFFT_SUCCESS);
		
		hipEventRecord(end_trans);
		hipEventSynchronize(end_trans);
		hipEventElapsedTime(&time, start_trans, end_trans);
		std::cout << "cheby fft s2p padding time = " << time / 1000.0 << std::endl;



		//err = hipDeviceSynchronize();
		//ASSERT(err == hipSuccess);
	}
	else if(doPadding == No_Padding){

		hipEventRecord(start_trans);
		cheby_pre_s2p_noPad<<<nBlock, nthread >>>((complex*)tPtr.ptr, tPtr.pitch, hmx, my, mz);
#ifdef KERNEL_SYNCHRONIZED
		err = hipDeviceSynchronize();
		assert(err == hipSuccess);
#endif
		hipEventRecord(end_trans);
		hipEventSynchronize(end_trans);
		hipEventElapsedTime(&time, start_trans, end_trans);
		std::cout << "cheby_pre_s2p_nopad_time = " << time / 1000.0 << std::endl;


		hipEventRecord(start_trans);
		res = CUFFTEXEC_C2C(planZ_no_pad, (CUFFTCOMPLEX*)tPtr.ptr,
			(CUFFTCOMPLEX*)tPtr.ptr, HIPFFT_FORWARD);
		ASSERT(res == HIPFFT_SUCCESS);
		
		hipEventRecord(end_trans);
		hipEventSynchronize(end_trans);
		hipEventElapsedTime(&time, start_trans, end_trans);
		std::cout << "cheby fft s2p no pad time = " << time / 1000.0 << std::endl;

		//err = hipDeviceSynchronize();
		//ASSERT(err == hipSuccess);
	}
	else
	{
		assert(false);		
	}
}

__host__ void transform_backward_X6(problem& pb) {
	int dim[3] = { pb.mx,pb.my,pb.mz };
	int tDim[3] = { pb.mz,pb.mx,pb.my };

	cheby_s2p_X6(pb.dptr_tu, dim[0] / 2 + 1, dim[1], dim[2]);

	//transpose(dir, Ptr, tPtr, dim, tDim);
	cuda_transpose(BACKWARD, pb.dptr_u, pb.dptr_tu, dim, tDim);
	cuda_transpose(BACKWARD, pb.dptr_v, pb.dptr_tv, dim, tDim);
	cuda_transpose(BACKWARD, pb.dptr_w, pb.dptr_tw, dim, tDim);
	cuda_transpose(BACKWARD, pb.dptr_omega_x, pb.dptr_tomega_x, dim, tDim);
	cuda_transpose(BACKWARD, pb.dptr_omega_y, pb.dptr_tomega_y, dim, tDim);
	cuda_transpose(BACKWARD, pb.dptr_omega_z, pb.dptr_tomega_z, dim, tDim);

	hipPitchedPtr& Ptr = pb.dptr_u;

	int nThreadx = 16;
	int nThready = 16;
	dim3 nThread(nThreadx, nThready);
	int nDimx = dim[1] / nThreadx;
	int nDimy = (dim[2] / 2 + 1)*6 / nThready;
	if (dim[1] % nThreadx != 0) nDimx++;
	if ((dim[2] / 2 + 1)*6 % nThready != 0) nDimy++;
	dim3 nDim(nDimx, nDimy);
	setZerosKernel<<<nDim, nThread >>>((complex*)Ptr.ptr, Ptr.pitch,
		dim[0], dim[1], dim[2]*6);
#ifdef KERNEL_SYNCHRONIZED
	cuCheck(hipDeviceSynchronize(), "set zeros");
#endif
	hipfftResult_t res;
	res = CUFFTEXEC_C2R(planXYc2r_X6, (CUFFTCOMPLEX*)pb.dptr_u.ptr,
		(CUFFTREAL*)pb.dptr_u.ptr);
	ASSERT(res == HIPFFT_SUCCESS);
	cuCheck(hipDeviceSynchronize(),"fft");
}

__host__ void transform_forward_X3(problem& pb) {
	hipfftResult_t res;
	hipPitchedPtr Ptr = pb.dptr_lamb_x;
	res = CUFFTEXEC_R2C(planXYr2c_X3, (CUFFTREAL*)Ptr.ptr,
		(CUFFTCOMPLEX*)Ptr.ptr);
	
	int dim[3] = { pb.mx, pb.my, pb.mz };
	int tDim[3] = { pb.mz, pb.mx, pb.my };
	//normalize;
	int nthreadx = 16;
	int nthready = 16;
	int nDimx = dim[1] / nthreadx;
	int nDimy = (dim[2] / 2 + 1) * 3/ nthready;
	if (dim[1] % nthreadx != 0) nDimx++;
	if ((dim[2] / 2 + 1)*3 % nthready != 0) nDimy++;
	dim3 dim_num(nDimx, nDimy);
	dim3 thread_num(nthreadx, nthready);

	// THIS LAUNCH PARAMETER NEED TO BE CHANGED
	normalizeKernel<<< dim_num, thread_num >>>
		(Ptr, dim[0], dim[1], dim[2]*3, 1.0 / dim[0] / dim[1]);
	cuCheck(hipDeviceSynchronize(),"normalize X3");


	cuda_transpose(FORWARD, pb.dptr_lamb_z, pb.dptr_tLamb_z, dim, tDim);
	cuda_transpose(FORWARD, pb.dptr_lamb_y, pb.dptr_tLamb_y, dim, tDim);
	cuda_transpose(FORWARD, pb.dptr_lamb_x, pb.dptr_tLamb_x, dim, tDim);
	
	cheby_p2s_X3(pb.dptr_tLamb_x, dim[0] / 2 + 1, dim[1], dim[2]);
}

__host__ void cheby_p2s_X3(hipPitchedPtr tPtr, int hmx, int my, int mz) {
//	const size_t pitch = tPtr.pitch;
	const int nx = (hmx - 1) * 2 / 3 * 2;
	const int ny = my / 3 * 2;
	const int hnx = nx / 2 + 1;

	int threadDimx = 16;
	int threadDimy = 16;

	int blockDimx = hnx / threadDimx;
	int blockDimy = ny*3 / threadDimy;

	if (hnx%threadDimx != 0) blockDimx++;
	if (ny*3%threadDimy != 0) blockDimy++;

	dim3 nthread(threadDimx, threadDimy);
	dim3 nBlock(blockDimx, blockDimy);

	hipfftResult res;
	hipError_t err;
	cheby_pre_p2s <<<nBlock, nthread >> >((complex*)tPtr.ptr, tPtr.pitch, hmx, my*3, mz);
#ifdef KERNEL_SYNCHRONIZED
	err = hipDeviceSynchronize();
	assert(err == hipSuccess);
#endif

	res = CUFFTEXEC_C2C(planZ_X3, (CUFFTCOMPLEX*)tPtr.ptr,
		(CUFFTCOMPLEX*)tPtr.ptr, HIPFFT_FORWARD);
	assert(res == HIPFFT_SUCCESS);

	//err = hipDeviceSynchronize();
	//assert(err == hipSuccess);

	cheby_post_p2s <<<nBlock, nthread >>>((complex*)tPtr.ptr, tPtr.pitch, hmx, my*3, mz);
#ifdef KERNEL_SYNCHRONIZED
	err = hipDeviceSynchronize();
	assert(err == hipSuccess);
#endif
}


__host__ void cheby_s2p_X6(hipPitchedPtr tPtr, int hmx, int my, int mz) {
//	const size_t pitch = tPtr.pitch;
	//const int pz = mz / 2 + 1;
	const int nx = (hmx - 1) * 2 / 3 * 2;
	const int ny = my / 3 * 2;
	const int hnx = nx / 2 + 1;

	int threadDimx = 16;
	int threadDimy = 16;

	int blockDimx = hnx / threadDimx;
	int blockDimy = 6*ny / threadDimy;

	if (hnx%threadDimx != 0) blockDimx++;
	if (6*ny%threadDimy != 0) blockDimy++;

	dim3 nthread(threadDimx, threadDimy);
	dim3 nBlock(blockDimx, blockDimy);
	hipfftResult res;
	hipError_t err;
	cheby_pre_s2p_pad <<<nBlock, nthread >>>((complex*)tPtr.ptr, tPtr.pitch, hmx, 6*my, mz);
#ifdef KERNEL_SYNCHRONIZED
	err = hipDeviceSynchronize();
	assert(err == hipSuccess);
#endif		

	res = CUFFTEXEC_C2C(planZ_X6, (CUFFTCOMPLEX*)tPtr.ptr,
		(CUFFTCOMPLEX*)tPtr.ptr, HIPFFT_FORWARD);
	ASSERT(res == HIPFFT_SUCCESS);

	//err = hipDeviceSynchronize();
	//ASSERT(err == hipSuccess);	
}