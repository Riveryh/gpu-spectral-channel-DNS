#include "hip/hip_runtime.h"
#include "transpose.cuh"
#include <cassert>
#include <iostream>
#include "cuRPCF.h"

#define TILE_DIM 8

__global__ void transpose_forward(cuRPCF::complex* u, cuRPCF::complex* tu, dim3 dim,
	size_t pitch, size_t tPitch);

__global__ void transpose_backward(cuRPCF::complex* u, cuRPCF::complex* tu, dim3 dim,
	size_t pitch, size_t tPitch);

__global__ void transpose_forward_sm(cuRPCF::complex* u, cuRPCF::complex* tu, dim3 dim,
	size_t pitch, size_t tPitch);

__global__ void transpose_backward_sm(cuRPCF::complex* u, cuRPCF::complex* tu, dim3 dim,
	size_t pitch, size_t tPitch);

// dim������ʾ����xyz���е����ݵ�ά�ȣ�Real��ʽ����ʵ��������transposeʱ�洢����Complex��ʽ��
// ���ʵ�ʵ�����ά��Ϊ 2*(dim[0]/2+1) x dim[1] x dim[2]
// tDim������ʾ����zxy���е����ݵ�ά�ȣ�Real��ʽ����ʵ��������transposeʱ�洢����Complex��ʽ��
// ���ʵ�ʵ�����ά��Ϊ dim[2] x 2*(dim[0]/2+1) x dim[1] 
// �� tDim[0] x (tDim[1]/2+1)*2 x tDim[2]
// ʵ����tDim������ʹ�ã�����������֤
__host__ int transpose(DIRECTION dir, hipPitchedPtr Ptr,
	hipPitchedPtr tPtr, int* dim, int* tDim) {
	//storage of host temporal variable
	cuRPCF::complex* buffer, *tbuffer;

	//number of cuRPCF::complex
	int nx = (dim[0] / 2 + 1);
	int ny = dim[1];
	int nz = dim[2];
	size_t Pitch = Ptr.pitch;
	size_t tPitch = tPtr.pitch;

	// sizes are defined in unit of bytes
	size_t size = Pitch * ny * nz;
	size_t tsize = tPitch * nx * ny;

	cuRPCF::complex* ptr = (cuRPCF::complex*)Ptr.ptr;
	cuRPCF::complex* tptr = (cuRPCF::complex*)tPtr.ptr;
	hipError_t err;

	//ASSERT(dir == BACKWARD);
	ASSERT(sizeof(cuRPCF::complex) == 2 * sizeof(REAL));
	ASSERT(dim[0] == tDim[1]);
	ASSERT(dim[1] == tDim[2]);
	ASSERT(dim[2] == tDim[0]);
	ASSERT(Pitch >= nx * sizeof(cuRPCF::complex));
	ASSERT(tPitch >= nz * sizeof(cuRPCF::complex));
	ASSERT(ptr != nullptr);
	ASSERT(tptr != nullptr);

	buffer = (cuRPCF::complex*)malloc(size);
	tbuffer = (cuRPCF::complex*)malloc(tsize);

	//set default value to zeros.
	for (size_t i = 0; i < size / sizeof(cuRPCF::complex); i++) {
		buffer[i] = 0.0;
	}
	for (size_t i = 0; i < tsize / sizeof(cuRPCF::complex); i++) {
		tbuffer[i] = 0.0;
	}

	size_t layerIn = Pitch / sizeof(cuRPCF::complex)*ny;
	// Ptr[z][y][x] = tPtr[y][x][z]
	if (dir == FORWARD) {
		err = hipMemcpy(buffer, ptr, size, hipMemcpyDeviceToHost);
		ASSERT(err == hipSuccess);
#pragma omp parallel for
		for (int k = 0; k < nz; k++) {
			for (int j = 0; j < ny; j++) {
				for (int i = 0; i < nx; i++) {
					size_t index1 = k*layerIn + Pitch / sizeof(cuRPCF::complex)*j + i;
					size_t index2 = (nx*j + i)*tPitch / sizeof(cuRPCF::complex) + k;
					tbuffer[index2] = buffer[index1];
				}
			}
		}
		err = hipMemcpy(tptr, tbuffer, tsize, hipMemcpyHostToDevice);
		ASSERT(err == hipSuccess);
		//err = hipDeviceSynchronize();
		//ASSERT(err == hipSuccess);
	}
	else
	{
		ASSERT(dir == BACKWARD);
		err = hipMemcpy(tbuffer, tptr, tsize, hipMemcpyDeviceToHost);
		ASSERT(err == hipSuccess);
		//hipDeviceSynchronize();
#pragma omp parallel for
		for (int k = 0; k < nz; k++) {
			for (int j = 0; j < ny; j++) {
				for (int i = 0; i < nx; i++) {
					size_t index1 = k*layerIn + Pitch / sizeof(cuRPCF::complex)*j + i;
					size_t index2 = (nx*j + i)*tPitch / sizeof(cuRPCF::complex) + k;
					buffer[index1] = tbuffer[index2];
				}
			}
		}
		err = hipMemcpy(ptr, buffer, size, hipMemcpyHostToDevice);
		ASSERT(hipSuccess == err);
	}
	//RPCF::write_3d_to_file("input.txt", (REAL*)buffer, Ptr.pitch,
	//	nx, ny, nz);
	//RPCF::write_3d_to_file("output.txt", (REAL*)tbuffer, tPtr.pitch,
	//	nz, nx, ny);
	free(buffer);
	free(tbuffer);
	return 0;
}

__host__ int cuda_transpose(DIRECTION dir, hipPitchedPtr& Ptr,
	hipPitchedPtr& tPtr, int* dim, int* tDim) {
	const int hnx = dim[0] / 3 * 2 / 2 + 1;
	const int ny = dim[1] / 3 * 2;
	const int mz = dim[2];
	int nthreadx = 16;
	int nthready = 16;
	
	dim3 dims(dim[0], dim[1], dim[2]);
	if (dir == FORWARD) {
		int nBlockx = hnx / nthreadx;
		int nBlocky = ny / nthready;
		if (hnx % nthreadx != 0) nBlockx++;
		if (ny % nthready != 0) nBlocky++;
		dim3 nBlock(nBlockx, nBlocky);
		dim3 nThread(nthreadx, nthready);

		//ASSERT(tPtr.ptr == nullptr);
		//cuCheck(hipMalloc3D(&(tPtr), tExtent),"cuMalloc");
		//cuCheck(myCudaMalloc(tPtr, ZXY_3D), "my hipMalloc");

		transpose_forward<<<dim3(hnx,ny),mz/2+1>>>((cuRPCF::complex*)Ptr.ptr, (cuRPCF::complex*)tPtr.ptr,
			dims, Ptr.pitch, tPtr.pitch);
		
		cuCheck(hipDeviceSynchronize(), "Transpose kernel");
		//cuCheck(myCudaFree(Ptr, XYZ_3D), "my cuda free at transform");
		//safeCudaFree(Ptr.ptr);
	}
	else if (dir == BACKWARD) {
		int nBlockx = dim[1] / nthreadx;
		int nBlocky = dim[2]/2+1 / nthready;
		if (dim[1] % nthreadx != 0) nBlockx++;
		if (dim[2]/2+1 % nthready != 0) nBlocky++;
		dim3 nBlock(nBlockx, nBlocky);
		dim3 nThread(nthreadx, nthready);

		//ASSERT(Ptr.ptr == nullptr);
		//cuCheck( hipMalloc3D(&(Ptr), pExtent),"cuMalloc");
		//cuCheck(myCudaMalloc(Ptr, XYZ_3D), "my hipMalloc");

		transpose_backward<<<dim3(hnx, ny), mz/2+1 >>>((cuRPCF::complex*)Ptr.ptr, (cuRPCF::complex*)tPtr.ptr,
			dims, Ptr.pitch, tPtr.pitch);
		cuCheck(hipDeviceSynchronize(), "Transpose kernel");

		//cuCheck(myCudaFree(tPtr, ZXY_3D), "my cuda free at transform");
		//safeCudaFree(tPtr.ptr);
	}
	else {
		std::cerr << "Wrong tranpose type!" << std::endl;
	}
	return 0;
}

__host__ int cuda_transpose_sm(DIRECTION dir, hipPitchedPtr& Ptr,
	hipPitchedPtr& tPtr, int* dim, int* tDim) {
	const int hnx = dim[0] / 3 * 2 / 2 + 1;
	const int ny = dim[1] / 3 * 2;
	const int mz = dim[2];
	const int pz = mz / 2 + 1;
	int nthreadx = TILE_DIM;
	int nthready = TILE_DIM;

	dim3 dims(dim[0], dim[1], dim[2]);
	if (dir == FORWARD) {
		//int max_length = hnx>pz ? hnx : pz;
		int nBlockx = hnx / TILE_DIM;
		int nBlocky = pz / TILE_DIM;
		if ((hnx % TILE_DIM) != 0) nBlockx++;
		if ((pz % TILE_DIM) != 0) nBlocky++;
		dim3 nBlock(nBlockx, nBlocky, ny);
		dim3 nThread(nthreadx, nthready);

		//ASSERT(tPtr.ptr == nullptr);
		//cuCheck(hipMalloc3D(&(tPtr), tExtent),"cuMalloc");
		//cuCheck(myCudaMalloc(tPtr, ZXY_3D), "my hipMalloc");

		transpose_forward_sm <<<nBlock, nThread>>>((cuRPCF::complex*)Ptr.ptr, (cuRPCF::complex*)tPtr.ptr,
			dims, Ptr.pitch, tPtr.pitch);

		cuCheck(hipDeviceSynchronize(), "Transpose kernel");
		//cuCheck(myCudaFree(Ptr, XYZ_3D), "my cuda free at transform");
		//safeCudaFree(Ptr.ptr);
	}
	else if (dir == BACKWARD) {
		int nBlockx = hnx / TILE_DIM;
		int nBlocky = pz / TILE_DIM;
		if ((hnx % TILE_DIM) != 0) nBlockx++;
		if ((pz % TILE_DIM) != 0) nBlocky++;
		dim3 nBlock(nBlockx, nBlocky, ny);
		dim3 nThread(nthreadx, nthready);

		//ASSERT(Ptr.ptr == nullptr);
		//cuCheck( hipMalloc3D(&(Ptr), pExtent),"cuMalloc");
		//cuCheck(myCudaMalloc(Ptr, XYZ_3D), "my hipMalloc");

		transpose_backward_sm <<<nBlock, nThread>>>((cuRPCF::complex*)Ptr.ptr, (cuRPCF::complex*)tPtr.ptr,
			dims, Ptr.pitch, tPtr.pitch);
		cuCheck(hipDeviceSynchronize(), "Transpose kernel");

		//cuCheck(myCudaFree(tPtr, ZXY_3D), "my cuda free at transform");
		//safeCudaFree(tPtr.ptr);
	}
	else {
		std::cerr << "Wrong tranpose type!" << std::endl;
	}
	return 0;
}

__global__ void transpose_forward(cuRPCF::complex* u, cuRPCF::complex* tu, dim3 dim,
	size_t pitch, size_t tPitch) {
	int kx = blockIdx.x;
	int ky = blockIdx.y;
	int kz = threadIdx.x;
	
	int mx = dim.x;
	int my = dim.y;
	int mz = dim.z;
	int nx = mx / 3 * 2;
	int ny = my / 3 * 2;
	int hnx = nx / 2 + 1;
	if (kx >= hnx) return;
	if (ky >= ny) return;
	if (kz >= mz/2+1)return;
	int old_ky = ky;
	int dky = my - ny;
	if (ky > ny / 2) old_ky = ky + dky;

	//for (int kz = 0; kz < mz/2+1; kz++) {
		size_t inc = pitch / sizeof(cuRPCF::complex)*(kz*my + old_ky) + kx;
		size_t tInc = tPitch / sizeof(cuRPCF::complex)*(ky*hnx + kx) + kz;
		tu[tInc] = u[inc];
	//}

	// NO NEED to set zeros here, 
	// because it will be covered by later setZero kernels.

	//if (ky == ny / 2 || kx == hnx - 1) {
	//	for (int kz = 0; kz < mz / 2 + 1; kz++) {
	//		size_t tInc = tPitch / sizeof(cuRPCF::complex)*(ky*hnx + kx) + kz;
	//		tu[tInc] = 0.0;
	//	}
	//}
}
__global__ void transpose_forward_sm(cuRPCF::complex* u, cuRPCF::complex* tu, dim3 dim,
	size_t pitch, size_t tPitch) {
	__shared__ cuRPCF::complex tile[TILE_DIM][TILE_DIM];

	int kx = blockIdx.x * TILE_DIM + threadIdx.x;
	int kz = blockIdx.y * TILE_DIM + threadIdx.y;
	const int ky = blockIdx.z;

	int mx = dim.x;
	int my = dim.y;
	int mz = dim.z;
	int ny = my / 3 * 2;
	int nx = mx / 3 * 2;
	int hnx = nx / 2 + 1;

	if (ky >= ny) return;

	const size_t size = pitch*my*(mz/2+1);
	const size_t tSize = tPitch*hnx*ny;

	int old_ky = ky;
	int dky = my - ny;
	if (ky > ny / 2) old_ky = ky + dky;

	if (kx < hnx) {
		//for (int iz = 0; iz < TILE_DIM && iz + kz<mz / 2 + 1; iz++) {
		if(kz < mz/2+1){
			size_t inc = pitch / sizeof(cuRPCF::complex)*(kz*my + old_ky) + kx;
			assert(inc * sizeof(cuRPCF::complex) < size);
			tile[threadIdx.y][threadIdx.x] = u[inc];
			//printf("block:%d,%d,%d,thread:%d,%d writing to: %x\n",blockIdx.x,blockIdx.y,blockIdx.z, threadIdx.x, threadIdx.y, inc);
		}
	}

	__syncthreads();

	kx = blockIdx.x * TILE_DIM + threadIdx.y;
	kz = blockIdx.y * TILE_DIM + threadIdx.x;


	if (kz < mz / 2 + 1) {
		//for (int ix = 0; ix < TILE_DIM && ix + kx<hnx; ix++) {
		if (kx < hnx){
			size_t tInc = tPitch / sizeof(cuRPCF::complex)*(ky*hnx + (kx)) + kz;
			assert(tInc * sizeof(cuRPCF::complex) < tSize);
			tu[tInc] = tile[threadIdx.x][threadIdx.y];
		}
	}
}

__global__ void transpose_backward_sm(cuRPCF::complex* u, cuRPCF::complex* tu, dim3 dim,
	size_t pitch, size_t tPitch) {
	__shared__ cuRPCF::complex tile[TILE_DIM][TILE_DIM];

	int kx = blockIdx.x * TILE_DIM + threadIdx.y;
	int kz = blockIdx.y * TILE_DIM + threadIdx.x;
	const int ky = blockIdx.z;

	int mx = dim.x;
	int my = dim.y;
	int mz = dim.z;
	int ny = my / 3 * 2;
	int nx = mx / 3 * 2;
	int hnx = nx / 2 + 1;

	if (ky >= ny) return;

	const size_t size = pitch*my*(mz / 2 + 1);
	const size_t tSize = tPitch*hnx*ny;

	int old_ky = ky;
	int dky = my - ny;
	if (ky > ny / 2) old_ky = ky + dky;

	if (kx < hnx) {
		//for (int iz = 0; iz < TILE_DIM && iz + kz<mz / 2 + 1; iz++) {
		if (kz < mz / 2 + 1) {
			size_t tInc = tPitch / sizeof(cuRPCF::complex)*(ky*hnx + (kx)) + kz;
			assert(tInc * sizeof(cuRPCF::complex) < tSize);
			tile[threadIdx.x][threadIdx.y] = tu[tInc];
			
			//printf("block:%d,%d,%d,thread:%d,%d writing to: %x\n",blockIdx.x,blockIdx.y,blockIdx.z, threadIdx.x, threadIdx.y, inc);
		}
	}

	__syncthreads();

	kx = blockIdx.x * TILE_DIM + threadIdx.x;
	kz = blockIdx.y * TILE_DIM + threadIdx.y;


	if (kz < mz / 2 + 1) {
		//for (int ix = 0; ix < TILE_DIM && ix + kx<hnx; ix++) {
		if (kx < hnx) {
			size_t inc = pitch / sizeof(cuRPCF::complex)*(kz*my + old_ky) + kx;
			assert(inc * sizeof(cuRPCF::complex) < size);
			u[inc] = tile[threadIdx.y][threadIdx.x];
		}
	}
}
__global__ void transpose_backward(cuRPCF::complex* u, cuRPCF::complex* tu, dim3 dim,
	size_t pitch, size_t tPitch) {
	int kx = blockIdx.x;
	int ky = blockIdx.y;
	int kz = threadIdx.x;

	int mx = dim.x;
	int my = dim.y;
	int mz = dim.z;
	int ny = my / 3 * 2;
	int nx = mx / 3 * 2;
	int hnx = nx / 2 + 1;
	if (kz >= mz / 2 + 1) return;
	if (ky >= ny) return;
	if (kx >= nx / 2 + 1)return;
	int old_ky = ky;
	int dky = my - ny;
	if (ky > ny / 2) old_ky = ky + dky;

	//for (int kx = 0; kx < nx/2+1; kx++) {
		size_t inc = pitch / sizeof(cuRPCF::complex)*(kz*my + old_ky) + kx;
		size_t tInc = tPitch / sizeof(cuRPCF::complex)*(ky*hnx + kx) + kz;
		u[inc] = tu[tInc];
	//}
}