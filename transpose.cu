#include "hip/hip_runtime.h"
#include "transpose.cuh"
#include <cassert>
#include <iostream>
#include "cuRPCF.h"

__global__ void transpose_forward(complex* u, complex* tu, dim3 dim,
	size_t pitch, size_t tPitch);

__global__ void transpose_backward(complex* u, complex* tu, dim3 dim,
	size_t pitch, size_t tPitch);

// dim������ʾ����xyz���е����ݵ�ά�ȣ�Real��ʽ����ʵ��������transposeʱ�洢����Complex��ʽ��
// ���ʵ�ʵ�����ά��Ϊ 2*(dim[0]/2+1) x dim[1] x dim[2]
// tDim������ʾ����zxy���е����ݵ�ά�ȣ�Real��ʽ����ʵ��������transposeʱ�洢����Complex��ʽ��
// ���ʵ�ʵ�����ά��Ϊ dim[2] x 2*(dim[0]/2+1) x dim[1] 
// �� tDim[0] x (tDim[1]/2+1)*2 x tDim[2]
// ʵ����tDim������ʹ�ã�����������֤
__host__ int transpose(DIRECTION dir, hipPitchedPtr Ptr,
	hipPitchedPtr tPtr, int* dim, int* tDim) {
	//storage of host temporal variable
	complex* buffer, *tbuffer;

	//number of complex
	int nx = (dim[0] / 2 + 1);
	int ny = dim[1];
	int nz = dim[2];
	size_t Pitch = Ptr.pitch;
	size_t tPitch = tPtr.pitch;

	// sizes are defined in unit of bytes
	size_t size = Pitch * ny * nz;
	size_t tsize = tPitch * nx * ny;

	complex* ptr = (complex*)Ptr.ptr;
	complex* tptr = (complex*)tPtr.ptr;
	hipError_t err;

	//ASSERT(dir == BACKWARD);
	ASSERT(sizeof(complex) == 2 * sizeof(real));
	ASSERT(dim[0] == tDim[1]);
	ASSERT(dim[1] == tDim[2]);
	ASSERT(dim[2] == tDim[0]);
	ASSERT(Pitch >= nx * sizeof(complex));
	ASSERT(tPitch >= nz * sizeof(complex));
	ASSERT(ptr != nullptr);
	ASSERT(tptr != nullptr);

	buffer = (complex*)malloc(size);
	tbuffer = (complex*)malloc(tsize);

	//set default value to zeros.
	for (size_t i = 0; i < size / sizeof(complex); i++) {
		buffer[i] = 0.0;
	}
	for (size_t i = 0; i < tsize / sizeof(complex); i++) {
		tbuffer[i] = 0.0;
	}

	size_t layerIn = Pitch / sizeof(complex)*ny;
	// Ptr[z][y][x] = tPtr[y][x][z]
	if (dir == FORWARD) {
		err = hipMemcpy(buffer, ptr, size, hipMemcpyDeviceToHost);
		ASSERT(err == hipSuccess);
#pragma omp parallel for
		for (int k = 0; k < nz; k++) {
			for (int j = 0; j < ny; j++) {
				for (int i = 0; i < nx; i++) {
					size_t index1 = k*layerIn + Pitch / sizeof(complex)*j + i;
					size_t index2 = (nx*j + i)*tPitch / sizeof(complex) + k;
					tbuffer[index2] = buffer[index1];
				}
			}
		}
		err = hipMemcpy(tptr, tbuffer, tsize, hipMemcpyHostToDevice);
		ASSERT(err == hipSuccess);
		//err = hipDeviceSynchronize();
		//ASSERT(err == hipSuccess);
	}
	else
	{
		ASSERT(dir == BACKWARD);
		err = hipMemcpy(tbuffer, tptr, tsize, hipMemcpyDeviceToHost);
		ASSERT(err == hipSuccess);
		//hipDeviceSynchronize();
#pragma omp parallel for
		for (int k = 0; k < nz; k++) {
			for (int j = 0; j < ny; j++) {
				for (int i = 0; i < nx; i++) {
					size_t index1 = k*layerIn + Pitch / sizeof(complex)*j + i;
					size_t index2 = (nx*j + i)*tPitch / sizeof(complex) + k;
					buffer[index1] = tbuffer[index2];
				}
			}
		}
		err = hipMemcpy(ptr, buffer, size, hipMemcpyHostToDevice);
		ASSERT(hipSuccess == err);
	}
	//RPCF::write_3d_to_file("input.txt", (real*)buffer, Ptr.pitch,
	//	nx, ny, nz);
	//RPCF::write_3d_to_file("output.txt", (real*)tbuffer, tPtr.pitch,
	//	nz, nx, ny);
	//investigate << <1, 1 >> > (Ptr);
	//investigate << <1, 1 >> > (tPtr);
	free(buffer);
	free(tbuffer);
	return 0;
}

__host__ int cuda_transpose(DIRECTION dir, hipPitchedPtr Ptr,
	hipPitchedPtr tPtr, int* dim, int* tDim) {
	int nthreadx = 16;
	int nthready = 16;
	
	dim3 dims(dim[0], dim[1], dim[2]);
	if (dir == FORWARD) {
		int nBlockx = dim[0] / nthreadx;
		int nBlocky = dim[1] / nthready;
		if (dim[0] % nthreadx != 0) nBlockx++;
		if (dim[1] % nthready != 0) nBlocky++;
		dim3 nBlock(nBlockx, nBlocky);
		dim3 nThread(nthreadx, nthready);

		transpose_forward<<<nBlock,nThread>>>((complex*)Ptr.ptr, (complex*)tPtr.ptr,
			dims, Ptr.pitch, tPtr.pitch);
		//DEBUG:cuCheck(hipDeviceSynchronize(),"Transpose kernel");
	}
	else if (dir == BACKWARD) {
		int nBlockx = dim[1] / nthreadx;
		int nBlocky = dim[2]/2+1 / nthready;
		if (dim[1] % nthreadx != 0) nBlockx++;
		if (dim[2]/2+1 % nthready != 0) nBlocky++;
		dim3 nBlock(nBlockx, nBlocky);
		dim3 nThread(nthreadx, nthready);

		transpose_backward<<<nBlock,nThread>>>((complex*)Ptr.ptr, (complex*)tPtr.ptr,
			dims, Ptr.pitch, tPtr.pitch);
		//DEBUG:cuCheck(hipDeviceSynchronize(), "Transpose kernel");
	}
	else {
		std::cerr << "Wrong tranpose type!" << std::endl;
	}
	return 0;
}

__global__ void transpose_forward(complex* u, complex* tu, dim3 dim,
	size_t pitch, size_t tPitch) {
	int kx = threadIdx.x + blockDim.x*blockIdx.x;
	int ky = threadIdx.y + blockDim.y*blockIdx.y;

	int mx = dim.x;
	int my = dim.y;
	int mz = dim.z;
	if (kx >= mx / 2 + 1) return;
	if (ky >= my) return;
	for (int kz = 0; kz < mz/2+1; kz++) {
		size_t inc = pitch / sizeof(complex)*(kz*my + ky) + kx;
		size_t tInc = tPitch / sizeof(complex)*(ky*((mx / 2) + 1) + kx) + kz;
		tu[tInc] = u[inc];
	}
}

__global__ void transpose_backward(complex* u, complex* tu, dim3 dim,
	size_t pitch, size_t tPitch) {
	int ky = threadIdx.x + blockDim.x*blockIdx.x;
	int kz = threadIdx.y + blockDim.y*blockIdx.y;

	int mx = dim.x;
	int my = dim.y;
	int mz = dim.z;
	if (kz >= mz / 2 + 1) return;
	if (ky >= my) return;
	for (int kx = 0; kx < mx/2+1; kx++) {
		size_t inc = pitch / sizeof(complex)*(kz*my + ky) + kx;
		size_t tInc = tPitch / sizeof(complex)*(ky*((mx / 2) + 1) + kx) + kz;
		u[inc] = tu[tInc];
	}
}